#include "hip/hip_runtime.h"
#include "gdiamond_gpu_naive_vectorized.cuh"
#include "gdiamond_gpu_mm_ver3.cuh"

int main(int argc, char* argv[]) {

  std::cerr << "mix mapping ver3->consider valley as mountain top.\n";
  std::cerr << "tiling parameters: BLT_MM_V3 = " << BLT_MM_V3 
            << ", NTX_MM_V3 = " << NTX_MM_V3 << ", NTY_MM_V3 = " << NTY_MM_V3 << ", NTZ_MM_V3 = " << NTZ_MM_V3 << "\n";
  std::cerr << "X dimension: MOUNTAIN_X_V3 = " << MOUNTAIN_X_V3 << ", VALLEY_X_V3 = " << VALLEY_X_V3 << "\n"; 
  std::cerr << "Y dimension: MOUNTAIN_Y_V3 = " << MOUNTAIN_Y_V3 << ", VALLEY_Y_V3 = " << VALLEY_Y_V3 << "\n"; 
  std::cerr << "Z dimension: MOUNTAIN_Z_V3 = " << MOUNTAIN_Z_V3 << ", VALLEY_Z_V3 = " << VALLEY_Z_V3 << "\n"; 
    
  if(argc != 5) {
    std::cerr << "usage: ./example/gpu_dt Tx Ty Tz num_timesteps\n";
    std::exit(EXIT_FAILURE);
  }

  // Tx Ty Tz are the number of tile stripes 
  // (mountain bottom + valley bottom, excluding the 1st mountain and the last valley)
  size_t Tx = std::atoi(argv[1]);
  size_t Ty = std::atoi(argv[2]);
  size_t Tz = std::atoi(argv[3]);
  size_t num_timesteps = std::atoi(argv[4]);

  size_t Nx = Tx * VALLEY_X_V3; 
  size_t Ny = Ty * VALLEY_Y_V3; 
  size_t Nz = Tz * VALLEY_Z_V3; 

  std::cout << "simulation space: Nx = " << Nx << ", Ny = " << Ny << ", Nz = " << Nz << "\n";
  gdiamond::gDiamond exp(Nx, Ny, Nz); 

  exp.update_FDTD_gpu_3D_warp_underutilization_fix_vectorized(num_timesteps); 
  // exp.update_FDTD_mix_mapping_gpu_ver3(num_timesteps, Tx, Ty, Tz);  

  // exp.print_results();

  // if(!exp.check_correctness_gpu()) {
  //   std::cerr << "results are wrong!\n";
  //   std::exit(EXIT_FAILURE);
  // }

  // std::cerr << "results are matched.\n";

  return 0;
}
