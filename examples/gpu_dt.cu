#include "hip/hip_runtime.h"
#include "gdiamond_gpu.cuh"

int main() {

  size_t Nx = 19;
  size_t Ny = 19;
  size_t Nz = 19;
  size_t num_timesteps = 4;
  gdiamond::gDiamond exp(Nx, Ny, Nz); 

  exp.update_FDTD_gpu_simulation_1_D(num_timesteps);

  std::cout << "\n\nshared_memory:\n";
  exp.update_FDTD_gpu_simulation_1_D_shmem(num_timesteps);

  // exp.print_results();

  if(!exp.check_correctness_simu_shmem()) {
    std::cerr << "error: results not match\n";
    std::exit(EXIT_FAILURE);
  }

  return 0;
}
