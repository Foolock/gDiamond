#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <chrono>

// handle errors in CUDA call
#define CUDACHECK(call)                                                        \
{                                                                          \
   const hipError_t error = call;                                         \
   if (error != hipSuccess)                                               \
   {                                                                       \
      printf("Error: %s:%d, ", __FILE__, __LINE__);                        \
      printf("code:%d, reason: %s\n", error, hipGetErrorString(error));   \
      exit(1);                                                             \
   }                                                                       \
} (void)0  // Ensures a semicolon is required after the macro call.

#define BLOCK_SIZE 1024

__global__ void update(float* Ex, float* Ey, float* Ez,
                  float* Hx, float* Hy, float* Hz,
                  float* Cax, float* Cay, float* Caz,
                  float* Cbx, float* Cby, float* Cbz,
                  float* Jx, float* Jy, float* Jz,
                  float dx,
                  size_t length
) {

  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if(tid < length) {
    Ex[tid] = Cax[tid]*Ex[tid] + Cbx[tid] * (Hz[tid] + Hy[tid] + Jx[tid]) * dx; 
    Ey[tid] = Cay[tid]*Ey[tid] + Cby[tid] * (Hx[tid] + Hz[tid] + Jy[tid]) * dx; 
    Ez[tid] = Caz[tid]*Ey[tid] + Cby[tid] * (Hx[tid] + Hz[tid] + Jy[tid]) * dx; 
  }

}

__global__ void check(float *Ex, 
                 size_t length
) {

  for(size_t i=0; i<length; i++) {
    printf("%f ", Ex[i]); 
  }
  printf("\n");

}

void sequential(std::vector<float>& Ex, std::vector<float>& Ey, std::vector<float>& Ez, 
                std::vector<float> Hx, std::vector<float> Hy, std::vector<float> Hz,
                std::vector<float> Cax, std::vector<float> Cay, std::vector<float> Caz,
                std::vector<float> Cbx, std::vector<float> Cby, std::vector<float> Cbz,
                std::vector<float> Jx, std::vector<float> Jy, std::vector<float> Jz,
                float dx,
                size_t length
) {

  auto start = std::chrono::steady_clock::now();
  
  for(size_t t=0; t<100; t++) {
    for(size_t i=0; i<length; i++) {
      Ex[i] = Cax[i]*Ex[i] + Cbx[i] * (Hz[i] + Hy[i] + Jx[i]) * dx; 
      Ey[i] = Cay[i]*Ey[i] + Cby[i] * (Hx[i] + Hz[i] + Jy[i]) * dx; 
      Ez[i] = Caz[i]*Ey[i] + Cby[i] * (Hx[i] + Hz[i] + Jy[i]) * dx; 
    }
  }

  auto end = std::chrono::steady_clock::now();
  size_t seq_runtime = std::chrono::duration_cast<std::chrono::microseconds>(end-start).count();
  std::cout << "seq runtime: " << seq_runtime << "us\n";
}

int main() {

  size_t Nx = 100;
  size_t Ny = 100;
  size_t Nz = 100;

  float dx = 0.01;

  std::vector<float> Ex(Nx * Ny * Nz, 1); 
  std::vector<float> Ey(Nx * Ny * Nz, 1); 
  std::vector<float> Ez(Nx * Ny * Nz, 1); 

  std::vector<float> Ex_c(Nx * Ny * Nz, 1); 
  std::vector<float> Ey_c(Nx * Ny * Nz, 1); 
  std::vector<float> Ez_c(Nx * Ny * Nz, 1); 

  std::vector<float> Hx(Nx * Ny * Nz, 1); 
  std::vector<float> Hy(Nx * Ny * Nz, 1); 
  std::vector<float> Hz(Nx * Ny * Nz, 1); 

  std::vector<float> Cax(Nx * Ny * Nz, 1); 
  std::vector<float> Cay(Nx * Ny * Nz, 1); 
  std::vector<float> Caz(Nx * Ny * Nz, 1); 

  std::vector<float> Cbx(Nx * Ny * Nz, 1); 
  std::vector<float> Cby(Nx * Ny * Nz, 1); 
  std::vector<float> Cbz(Nx * Ny * Nz, 1); 

  std::vector<float> Jx(Nx * Ny * Nz, 1); 
  std::vector<float> Jy(Nx * Ny * Nz, 1); 
  std::vector<float> Jz(Nx * Ny * Nz, 1); 

  sequential(Ex_c, Ey_c, Ez_c, 
             Hx, Hy, Hz,
             Cax, Cay, Caz,
             Cbx, Cby, Cbz,
             Jx, Jy, Jz,
             dx,
             Nx*Ny*Nz);

  float *Ex_d, *Ey_d, *Ez_d;
  float *Hx_d, *Hy_d, *Hz_d;
  float *Cax_d, *Cay_d, *Caz_d;
  float *Cbx_d, *Cby_d, *Cbz_d;
  float *Jx_d, *Jy_d, *Jz_d;

  CUDACHECK(hipMalloc(&Ex_d, sizeof(float) * Nx * Ny * Nz));
  CUDACHECK(hipMalloc(&Ey_d, sizeof(float) * Nx * Ny * Nz));
  CUDACHECK(hipMalloc(&Ez_d, sizeof(float) * Nx * Ny * Nz));
  CUDACHECK(hipMalloc(&Hx_d, sizeof(float) * Nx * Ny * Nz));
  CUDACHECK(hipMalloc(&Hy_d, sizeof(float) * Nx * Ny * Nz));
  CUDACHECK(hipMalloc(&Hz_d, sizeof(float) * Nx * Ny * Nz));
  CUDACHECK(hipMalloc(&Cax_d, sizeof(float) * Nx * Ny * Nz));
  CUDACHECK(hipMalloc(&Cay_d, sizeof(float) * Nx * Ny * Nz));
  CUDACHECK(hipMalloc(&Caz_d, sizeof(float) * Nx * Ny * Nz));
  CUDACHECK(hipMalloc(&Cbx_d, sizeof(float) * Nx * Ny * Nz));
  CUDACHECK(hipMalloc(&Cby_d, sizeof(float) * Nx * Ny * Nz));
  CUDACHECK(hipMalloc(&Cbz_d, sizeof(float) * Nx * Ny * Nz));
  CUDACHECK(hipMalloc(&Jx_d, sizeof(float) * Nx * Ny * Nz));
  CUDACHECK(hipMalloc(&Jy_d, sizeof(float) * Nx * Ny * Nz));
  CUDACHECK(hipMalloc(&Jz_d, sizeof(float) * Nx * Ny * Nz));

  hipMemcpy(Ex_d, Ex.data(), sizeof(float) * Nx * Ny * Nz, hipMemcpyHostToDevice);
  hipMemcpy(Ey_d, Ey.data(), sizeof(float) * Nx * Ny * Nz, hipMemcpyHostToDevice);
  hipMemcpy(Ez_d, Ez.data(), sizeof(float) * Nx * Ny * Nz, hipMemcpyHostToDevice);
  hipMemcpy(Hx_d, Hx.data(), sizeof(float) * Nx * Ny * Nz, hipMemcpyHostToDevice);
  hipMemcpy(Hy_d, Hy.data(), sizeof(float) * Nx * Ny * Nz, hipMemcpyHostToDevice);
  hipMemcpy(Hz_d, Hz.data(), sizeof(float) * Nx * Ny * Nz, hipMemcpyHostToDevice);
  hipMemcpy(Jx_d, Jx.data(), sizeof(float) * Nx * Ny * Nz, hipMemcpyHostToDevice);
  hipMemcpy(Jy_d, Jy.data(), sizeof(float) * Nx * Ny * Nz, hipMemcpyHostToDevice);
  hipMemcpy(Jz_d, Jz.data(), sizeof(float) * Nx * Ny * Nz, hipMemcpyHostToDevice);

  auto start = std::chrono::steady_clock::now();

  hipMemcpy(Cax_d, Cax.data(), sizeof(float) * Nx * Ny * Nz, hipMemcpyHostToDevice);
  hipMemcpy(Cay_d, Cay.data(), sizeof(float) * Nx * Ny * Nz, hipMemcpyHostToDevice);
  hipMemcpy(Caz_d, Caz.data(), sizeof(float) * Nx * Ny * Nz, hipMemcpyHostToDevice);
  hipMemcpy(Cbx_d, Cbx.data(), sizeof(float) * Nx * Ny * Nz, hipMemcpyHostToDevice);
  hipMemcpy(Cby_d, Cby.data(), sizeof(float) * Nx * Ny * Nz, hipMemcpyHostToDevice);
  hipMemcpy(Cbz_d, Cbz.data(), sizeof(float) * Nx * Ny * Nz, hipMemcpyHostToDevice);

  size_t grid_size = (Nx*Ny*Nz + BLOCK_SIZE - 1)/BLOCK_SIZE;

  for(size_t t=0; t<100; t++) {
    update<<<grid_size, BLOCK_SIZE>>>(Ex_d, Ey_d, Ez_d,
                                      Hx_d, Hy_d, Hz_d,
                                      Cax_d, Cay_d, Caz_d,
                                      Cbx_d, Cby_d, Cbz_d,
                                      Jx_d, Jy_d, Jz_d,
                                      dx,
                                      Nx*Ny*Nz
                                     );

  }

  hipMemcpy(Ex.data(), Ex_d, sizeof(float) * Nx * Ny * Nz, hipMemcpyDeviceToHost);
  hipMemcpy(Ey.data(), Ey_d, sizeof(float) * Nx * Ny * Nz, hipMemcpyDeviceToHost);
  hipMemcpy(Ez.data(), Ez_d, sizeof(float) * Nx * Ny * Nz, hipMemcpyDeviceToHost);

  auto end = std::chrono::steady_clock::now();
  size_t gpu_runtime = std::chrono::duration_cast<std::chrono::microseconds>(end-start).count();
  std::cout << "gpu runtime: " << gpu_runtime << "us\n";

  bool correct = true;
  for(size_t i=0; i<Nx*Ny*Nz; i++) {
    if(fabs(Ex_c[i] - Ex[i]) > 1e-5 ||
       fabs(Ey_c[i] - Ey[i]) > 1e-5 ||
       fabs(Ez_c[i] - Ez[i]) > 1e-5) {
      correct = false;
      break;
    }
  }
  if(!correct) {
    std::cout << "results incorrect!\n";
  }

  // std::cout << "Ex_c = \n";
  // for(size_t i=0; i<Nx*Ny*Nz; i++) {
  //   std::cout << Ex_c[i] << " ";
  // }
  // std::cout << "\n";

  // std::cout << "Ex = \n";
  // for(size_t i=0; i<Nx*Ny*Nz; i++) {
  //   std::cout << Ex[i] << " ";
  // }
  // std::cout << "\n";

  hipFree(Ex_d);
  hipFree(Ey_d);
  hipFree(Ez_d);
  hipFree(Hx_d);
  hipFree(Hy_d);
  hipFree(Hz_d);
  hipFree(Cax_d);
  hipFree(Cay_d);
  hipFree(Caz_d);
  hipFree(Cbx_d);
  hipFree(Cby_d);
  hipFree(Cbz_d);
  hipFree(Jx_d);

  hipFree(Ex_d);
  hipFree(Ey_d);
  hipFree(Ez_d);
  hipFree(Hx_d);
  hipFree(Hy_d);
  hipFree(Hz_d);
  hipFree(Cax_d);
  hipFree(Cay_d);
  hipFree(Caz_d);
  hipFree(Cbx_d);
  hipFree(Cby_d);
  hipFree(Cbz_d);
  hipFree(Jx_d);
  hipFree(Jy_d);
  hipFree(Jz_d);

  return 0;
}

























