#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN

#include <doctest.h>
#include "gdiamond_gpu.cuh"

// --------------------------------------------------------
// Testcase: check correctness of gpu compared to single thread implementation 
// --------------------------------------------------------
TEST_CASE("check correctness of gpu" * doctest::timeout(300)) {

  size_t Nx = 100;
  size_t Ny = 100;
  size_t Nz = 100;
  size_t num_timesteps = 100;
  gdiamond::gDiamond exp(Nx, Ny, Nz); 

  exp.update_FDTD_seq(num_timesteps);
  exp.update_FDTD_gpu_3D_warp_underutilization_fix(num_timesteps);

  REQUIRE(exp.check_correctness_gpu() == true);

}


