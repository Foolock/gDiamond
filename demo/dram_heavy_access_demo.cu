#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <chrono>

// handle errors in CUDA call
#define CUDACHECK(call)                                                        \
{                                                                          \
   const hipError_t error = call;                                         \
   if (error != hipSuccess)                                               \
   {                                                                       \
      printf("Error: %s:%d, ", __FILE__, __LINE__);                        \
      printf("code:%d, reason: %s\n", error, hipGetErrorString(error));   \
      exit(1);                                                             \
   }                                                                       \
} (void)0  // Ensures a semicolon is required after the macro call.

#define BLOCK_SIZE 1024

__global__ void update(float* Ex, float* Ey, float* Ez,
                  float* Hx, float* Hy, float* Hz,
                  float* Cax, float* Cay, float* Caz,
                  float* Cbx, float* Cby, float* Cbz,
                  float* Jx, float* Jy, float* Jz,
                  float dx,
                  size_t length
) {

  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if(tid < length) {
    Ex[tid] = Cax[tid]*Ex[tid] + Cbx[tid] * (Hz[tid] + Hy[tid] + Jx[tid]) * dx; 
    Ey[tid] = Cay[tid]*Ey[tid] + Cby[tid] * (Hx[tid] + Hz[tid] + Jy[tid]) * dx; 
    Ez[tid] = Caz[tid]*Ey[tid] + Cby[tid] * (Hx[tid] + Hz[tid] + Jy[tid]) * dx; 
  }

}

__global__ void update_half_dram_access(float* Ex, float* Ey, float* Ez,
                  float* Cax, float* Cay, float* Caz,
                  float* Cbx, float* Cby, float* Cbz,
                  size_t length
) {

  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if(tid < length) {
    Ex[tid] = Cax[tid]*Ex[tid] + Cbx[tid] * (1 + 1 + 1) * 0.01; 
    Ey[tid] = Cay[tid]*Ey[tid] + Cby[tid] * (1 + 1 + 1) * 0.01; 
    Ez[tid] = Caz[tid]*Ey[tid] + Cby[tid] * (1 + 1 + 1) * 0.01; 
  }

}

__global__ void check(float *Ex, 
                 size_t length
) {

  for(size_t i=0; i<length; i++) {
    printf("%f ", Ex[i]); 
  }
  printf("\n");

}

void sequential(std::vector<float>& Ex, std::vector<float>& Ey, std::vector<float>& Ez, 
                std::vector<float> Hx, std::vector<float> Hy, std::vector<float> Hz,
                std::vector<float> Cax, std::vector<float> Cay, std::vector<float> Caz,
                std::vector<float> Cbx, std::vector<float> Cby, std::vector<float> Cbz,
                std::vector<float> Jx, std::vector<float> Jy, std::vector<float> Jz,
                float dx,
                size_t length
) {

  auto start = std::chrono::steady_clock::now();
  
  for(size_t t=0; t<100; t++) {
    for(size_t i=0; i<length; i++) {
      Ex[i] = Cax[i]*Ex[i] + Cbx[i] * (Hz[i] + Hy[i] + Jx[i]) * dx; 
      Ey[i] = Cay[i]*Ey[i] + Cby[i] * (Hx[i] + Hz[i] + Jy[i]) * dx; 
      Ez[i] = Caz[i]*Ey[i] + Cby[i] * (Hx[i] + Hz[i] + Jy[i]) * dx; 
    }
  }

  auto end = std::chrono::steady_clock::now();
  size_t seq_runtime = std::chrono::duration_cast<std::chrono::microseconds>(end-start).count();
  std::cout << "seq runtime: " << seq_runtime << "us\n";
}

void sequential_half_dram_access(std::vector<float>& Ex, std::vector<float>& Ey, std::vector<float>& Ez, 
                std::vector<float> Cax, std::vector<float> Cay, std::vector<float> Caz,
                std::vector<float> Cbx, std::vector<float> Cby, std::vector<float> Cbz,
                size_t length
) {

  auto start = std::chrono::steady_clock::now();
  
  for(size_t t=0; t<100; t++) {
    for(size_t i=0; i<length; i++) {
      Ex[i] = Cax[i]*Ex[i] + Cbx[i] * (1 + 1 + 1) * 0.01; 
      Ey[i] = Cay[i]*Ey[i] + Cby[i] * (1 + 1 + 1) * 0.01; 
      Ez[i] = Caz[i]*Ey[i] + Cby[i] * (1 + 1 + 1) * 0.01; 
    }
  }

  auto end = std::chrono::steady_clock::now();
  size_t seq_runtime = std::chrono::duration_cast<std::chrono::microseconds>(end-start).count();
  std::cout << "seq runtime (half_dram_access): " << seq_runtime << "us\n";
}

void gpu_half_dram_access(std::vector<float>& Ex, std::vector<float>& Ey, std::vector<float>& Ez, 
         std::vector<float> Hx, std::vector<float> Hy, std::vector<float> Hz,
         std::vector<float> Cax, std::vector<float> Cay, std::vector<float> Caz,
         std::vector<float> Cbx, std::vector<float> Cby, std::vector<float> Cbz,
         std::vector<float> Jx, std::vector<float> Jy, std::vector<float> Jz,
         float dx,
         size_t length
) {

  float *Ex_d, *Ey_d, *Ez_d;
  float *Hx_d, *Hy_d, *Hz_d;
  float *Cax_d, *Cay_d, *Caz_d;
  float *Cbx_d, *Cby_d, *Cbz_d;
  float *Jx_d, *Jy_d, *Jz_d;

  CUDACHECK(hipMalloc(&Ex_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Ey_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Ez_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Hx_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Hy_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Hz_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Cax_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Cay_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Caz_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Cbx_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Cby_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Cbz_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Jx_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Jy_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Jz_d, sizeof(float) * length));

  hipMemcpy(Ex_d, Ex.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Ey_d, Ey.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Ez_d, Ez.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Hx_d, Hx.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Hy_d, Hy.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Hz_d, Hz.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Jx_d, Jx.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Jy_d, Jy.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Jz_d, Jz.data(), sizeof(float) * length, hipMemcpyHostToDevice);

  auto start = std::chrono::steady_clock::now();

  hipMemcpy(Cax_d, Cax.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Cay_d, Cay.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Caz_d, Caz.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Cbx_d, Cbx.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Cby_d, Cby.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Cbz_d, Cbz.data(), sizeof(float) * length, hipMemcpyHostToDevice);

  size_t grid_size = (length + BLOCK_SIZE - 1)/BLOCK_SIZE;

  for(size_t t=0; t<100; t++) {
    update_half_dram_access<<<grid_size, BLOCK_SIZE>>>(Ex_d, Ey_d, Ez_d,
                                                       Cax_d, Cay_d, Caz_d,
                                                       Cbx_d, Cby_d, Cbz_d,
                                                       length 
                                                      );

  }

  hipMemcpy(Ex.data(), Ex_d, sizeof(float) * length, hipMemcpyDeviceToHost);
  hipMemcpy(Ey.data(), Ey_d, sizeof(float) * length, hipMemcpyDeviceToHost);
  hipMemcpy(Ez.data(), Ez_d, sizeof(float) * length, hipMemcpyDeviceToHost);

  auto end = std::chrono::steady_clock::now();
  size_t gpu_runtime = std::chrono::duration_cast<std::chrono::microseconds>(end-start).count();
  std::cout << "gpu runtime ((half_dram_access)): " << gpu_runtime << "us\n";

  hipFree(Ex_d);
  hipFree(Ey_d);
  hipFree(Ez_d);
  hipFree(Hx_d);
  hipFree(Hy_d);
  hipFree(Hz_d);
  hipFree(Cax_d);
  hipFree(Cay_d);
  hipFree(Caz_d);
  hipFree(Cbx_d);
  hipFree(Cby_d);
  hipFree(Cbz_d);
  hipFree(Jx_d);
  hipFree(Jy_d);
  hipFree(Jz_d);
}

void gpu(std::vector<float>& Ex, std::vector<float>& Ey, std::vector<float>& Ez, 
         std::vector<float> Hx, std::vector<float> Hy, std::vector<float> Hz,
         std::vector<float> Cax, std::vector<float> Cay, std::vector<float> Caz,
         std::vector<float> Cbx, std::vector<float> Cby, std::vector<float> Cbz,
         std::vector<float> Jx, std::vector<float> Jy, std::vector<float> Jz,
         float dx,
         size_t length
) {

  float *Ex_d, *Ey_d, *Ez_d;
  float *Hx_d, *Hy_d, *Hz_d;
  float *Cax_d, *Cay_d, *Caz_d;
  float *Cbx_d, *Cby_d, *Cbz_d;
  float *Jx_d, *Jy_d, *Jz_d;

  CUDACHECK(hipMalloc(&Ex_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Ey_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Ez_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Hx_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Hy_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Hz_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Cax_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Cay_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Caz_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Cbx_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Cby_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Cbz_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Jx_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Jy_d, sizeof(float) * length));
  CUDACHECK(hipMalloc(&Jz_d, sizeof(float) * length));

  hipMemcpy(Ex_d, Ex.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Ey_d, Ey.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Ez_d, Ez.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Hx_d, Hx.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Hy_d, Hy.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Hz_d, Hz.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Jx_d, Jx.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Jy_d, Jy.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Jz_d, Jz.data(), sizeof(float) * length, hipMemcpyHostToDevice);

  auto start = std::chrono::steady_clock::now();

  hipMemcpy(Cax_d, Cax.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Cay_d, Cay.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Caz_d, Caz.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Cbx_d, Cbx.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Cby_d, Cby.data(), sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(Cbz_d, Cbz.data(), sizeof(float) * length, hipMemcpyHostToDevice);

  size_t grid_size = (length + BLOCK_SIZE - 1)/BLOCK_SIZE;

  for(size_t t=0; t<100; t++) {
    update<<<grid_size, BLOCK_SIZE>>>(Ex_d, Ey_d, Ez_d,
                                      Hx_d, Hy_d, Hz_d,
                                      Cax_d, Cay_d, Caz_d,
                                      Cbx_d, Cby_d, Cbz_d,
                                      Jx_d, Jy_d, Jz_d,
                                      dx,
                                      length 
                                     );

  }

  hipMemcpy(Ex.data(), Ex_d, sizeof(float) * length, hipMemcpyDeviceToHost);
  hipMemcpy(Ey.data(), Ey_d, sizeof(float) * length, hipMemcpyDeviceToHost);
  hipMemcpy(Ez.data(), Ez_d, sizeof(float) * length, hipMemcpyDeviceToHost);

  auto end = std::chrono::steady_clock::now();
  size_t gpu_runtime = std::chrono::duration_cast<std::chrono::microseconds>(end-start).count();
  std::cout << "gpu runtime: " << gpu_runtime << "us\n";

  hipFree(Ex_d);
  hipFree(Ey_d);
  hipFree(Ez_d);
  hipFree(Hx_d);
  hipFree(Hy_d);
  hipFree(Hz_d);
  hipFree(Cax_d);
  hipFree(Cay_d);
  hipFree(Caz_d);
  hipFree(Cbx_d);
  hipFree(Cby_d);
  hipFree(Cbz_d);
  hipFree(Jx_d);
  hipFree(Jy_d);
  hipFree(Jz_d);
}

int main() {

  size_t Nx = 100;
  size_t Ny = 100;
  size_t Nz = 100;

  float dx = 0.01;

  std::vector<float> Ex(Nx * Ny * Nz, 1); 
  std::vector<float> Ey(Nx * Ny * Nz, 1); 
  std::vector<float> Ez(Nx * Ny * Nz, 1); 

  std::vector<float> Ex_c(Nx * Ny * Nz, 1); 
  std::vector<float> Ey_c(Nx * Ny * Nz, 1); 
  std::vector<float> Ez_c(Nx * Ny * Nz, 1); 

  std::vector<float> Ex_c1(Nx * Ny * Nz, 1); 
  std::vector<float> Ey_c1(Nx * Ny * Nz, 1); 
  std::vector<float> Ez_c1(Nx * Ny * Nz, 1); 

  std::vector<float> Ex_g(Nx * Ny * Nz, 1); 
  std::vector<float> Ey_g(Nx * Ny * Nz, 1); 
  std::vector<float> Ez_g(Nx * Ny * Nz, 1); 

  std::vector<float> Ex_g1(Nx * Ny * Nz, 1); 
  std::vector<float> Ey_g1(Nx * Ny * Nz, 1); 
  std::vector<float> Ez_g1(Nx * Ny * Nz, 1); 

  std::vector<float> Hx(Nx * Ny * Nz, 1); 
  std::vector<float> Hy(Nx * Ny * Nz, 1); 
  std::vector<float> Hz(Nx * Ny * Nz, 1); 

  std::vector<float> Cax(Nx * Ny * Nz, 1); 
  std::vector<float> Cay(Nx * Ny * Nz, 1); 
  std::vector<float> Caz(Nx * Ny * Nz, 1); 

  std::vector<float> Cbx(Nx * Ny * Nz, 1); 
  std::vector<float> Cby(Nx * Ny * Nz, 1); 
  std::vector<float> Cbz(Nx * Ny * Nz, 1); 

  std::vector<float> Jx(Nx * Ny * Nz, 1); 
  std::vector<float> Jy(Nx * Ny * Nz, 1); 
  std::vector<float> Jz(Nx * Ny * Nz, 1); 

  sequential(Ex_c, Ey_c, Ez_c, 
             Hx, Hy, Hz,
             Cax, Cay, Caz,
             Cbx, Cby, Cbz,
             Jx, Jy, Jz,
             dx,
             Nx*Ny*Nz);

  sequential_half_dram_access(Ex_c1, Ey_c1, Ez_c1, 
                              Cax, Cay, Caz,
                              Cbx, Cby, Cbz,
                              Nx*Ny*Nz);

  gpu(Ex_g, Ey_g, Ez_g, 
      Hx, Hy, Hz,
      Cax, Cay, Caz,
      Cbx, Cby, Cbz,
      Jx, Jy, Jz,
      dx,
      Nx*Ny*Nz);

  gpu_half_dram_access(Ex_g1, Ey_g1, Ez_g1, 
      Hx, Hy, Hz,
      Cax, Cay, Caz,
      Cbx, Cby, Cbz,
      Jx, Jy, Jz,
      dx,
      Nx*Ny*Nz);

  bool correct = true;
  for(size_t i=0; i<Nx*Ny*Nz; i++) {
    if(fabs(Ex_c[i] - Ex_g[i]) > 1e-5 ||
       fabs(Ey_c[i] - Ey_g[i]) > 1e-5 ||
       fabs(Ez_c[i] - Ez_g[i]) > 1e-5) {
      correct = false;
      break;
    }
  }
  for(size_t i=0; i<Nx*Ny*Nz; i++) {
    if(fabs(Ex_c1[i] - Ex_g1[i]) > 1e-5 ||
       fabs(Ey_c1[i] - Ey_g1[i]) > 1e-5 ||
       fabs(Ez_c1[i] - Ez_g1[i]) > 1e-5) {
      correct = false;
      break;
    }
  }

  if(!correct) {
    std::cout << "results incorrect!\n";
  }

  return 0;
}

























